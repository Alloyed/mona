#include "hip/hip_runtime.h"
/* from dhowey, howey on github */
#include <hip/hip_runtime.h>
#include <cairo.h>
#include <math.h>
#include "mona.h"

extern "C" {
	int difference(cairo_surface_t * test_surf, cairo_surface_t * goal_surf);
	int get_max_fitness(void);
}
//The CUDA block size
#define BLOCK_SIZE 16

int MAX_FITNESS = -1;

unsigned char * goal_data = NULL;
unsigned char * goal_data_d;

__global__ void differenceKernel(unsigned char * test_data, unsigned char * goal_data, int * difference, int * my_max_fitness, int width, int height)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tx * width + ty;

    int difference_s = 0;
    int my_max_fitness_s = 0;

    if(i < height*width) {
	    int thispixel = 4 * i;
	    //int thispixel = tx*WIDTH*4 + ty*4;

	    unsigned char test_a = test_data[thispixel];
	    unsigned char test_r = test_data[thispixel + 1];
	    unsigned char test_g = test_data[thispixel + 2];
	    unsigned char test_b = test_data[thispixel + 3];

            unsigned char goal_a = goal_data[thispixel];
	    unsigned char goal_r = goal_data[thispixel + 1];
	    unsigned char goal_g = goal_data[thispixel + 2];
	    unsigned char goal_b = goal_data[thispixel + 3];

	    my_max_fitness_s += goal_a + goal_r + goal_g + goal_b;

	    difference_s += (ABS(test_a - goal_a) + ABS(test_r - goal_r) + ABS(test_g - goal_g) + ABS(test_b - goal_b));
    }

    my_max_fitness[i] = my_max_fitness_s;
    difference[i] = difference_s;
}

int difference(cairo_surface_t * test_surf, cairo_surface_t * goal_surf)
{
    unsigned char * test_data = cairo_image_surface_get_data(test_surf);
    if(!goal_data) {
        goal_data = cairo_image_surface_get_data(goal_surf);
    	hipMemcpy(goal_data_d, goal_data, sizeof(unsigned char)*4*WIDTH*HEIGHT, hipMemcpyHostToDevice);
    }

    unsigned char * test_data_d;
    int * difference;
    int * my_max_fitness;
    int * difference_d;
    int * my_max_fitness_d;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim(ceil((float)WIDTH/(float)BLOCK_SIZE), ceil((float)HEIGHT/(float)BLOCK_SIZE), 1);

    //TODO: Make these pointers global and only malloc once during the entire program
    hipMalloc((void **)&test_data_d, sizeof(unsigned char)*4*WIDTH*HEIGHT);
    hipMalloc((void **)&difference_d, sizeof(int)*WIDTH*HEIGHT);
    hipMalloc((void **)&my_max_fitness_d, sizeof(int)*WIDTH*HEIGHT);
    difference = (int *)malloc(sizeof(int)*WIDTH*HEIGHT);
    my_max_fitness = (int *)malloc(sizeof(int)*WIDTH*HEIGHT);

    //This will really slow things down. PCI-E bus will be a bottleneck.
    hipMemcpy(test_data_d, test_data, sizeof(unsigned char)*4*WIDTH*HEIGHT, hipMemcpyHostToDevice);

    //Launch the kernel to compute the difference
    differenceKernel<<<gridDim, blockDim>>>(test_data_d, goal_data_d, difference_d, my_max_fitness_d, WIDTH, HEIGHT);

    //Copy results from the device, another PCI-E bottleneck
    hipMemcpy(difference, difference_d, sizeof(int)*WIDTH*HEIGHT, hipMemcpyDeviceToHost);
    hipMemcpy(my_max_fitness, my_max_fitness_d, sizeof(int)*WIDTH*HEIGHT, hipMemcpyDeviceToHost);

    /*
    int difference = 0;

    int my_max_fitness = 0;

    #pragma omp parallel for 
    for(int y = 0; y < HEIGHT; y++)
    {
        for(int x = 0; x < WIDTH; x++)
        {
            int thispixel = y*WIDTH*4 + x*4;

            unsigned char test_a = test_data[thispixel];
            unsigned char test_r = test_data[thispixel + 1];
            unsigned char test_g = test_data[thispixel + 2];
            unsigned char test_b = test_data[thispixel + 3];

            unsigned char goal_a = goal_data[thispixel];
            unsigned char goal_r = goal_data[thispixel + 1];
            unsigned char goal_g = goal_data[thispixel + 2];
            unsigned char goal_b = goal_data[thispixel + 3];

            if(MAX_FITNESS == -1)
                my_max_fitness += goal_a + goal_r + goal_g + goal_b;

		#pragma omp atomic
	    difference += (ABS(test_a - goal_a) + ABS(test_r - goal_r) + ABS(test_g - goal_g) + ABS(test_b - goal_b));
        }
    }
    */

    //TODO: perform reduction on the GPU. Probalby won't be much speedup anyways
    int my_max_fitness_total = 0;
    int difference_total = 0;

    for(int i = 0; i < WIDTH*HEIGHT; i++) {
	    my_max_fitness_total += my_max_fitness[i];
	    difference_total += difference[i];
    }

    if(MAX_FITNESS == -1)
        MAX_FITNESS = my_max_fitness_total;

    hipFree(test_data_d);
    hipFree(difference_d);
    hipFree(my_max_fitness_d);
    free(difference);
    free(my_max_fitness);

    return difference_total;
}

int get_max_fitness()
{
	return MAX_FITNESS;
}
